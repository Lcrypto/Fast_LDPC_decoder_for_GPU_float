#include "hip/hip_runtime.h"
/*
 *  ldcp_decoder.h
 *  ldpc3
 *
 *  Created by legal on 02/04/11.
 *  Copyright 2011 ENSEIRB. All rights reserved.
 *
 */

/*----------------------------------------------------------------------------*/

#include "CGPU_Decoder_SCMS.h"
#include "../transpose/GPU_Transpose.h"

CGPU_Decoder_SCMS::CGPU_Decoder_SCMS(int _nb_frames, int block_size, unsigned int n, unsigned int k, unsigned int m):
CGPUDecoder(_nb_frames, block_size, n, k, m)
{
	size_t nb_blocks = nb_frames / BLOCK_SIZE;
	printf("(II) Decoder configuration: BLOCK_SIZE = %ld, nb_frames = %ld, nb_blocks = %ld\n", BLOCK_SIZE, nb_frames, nb_blocks);

	struct hipDeviceProp_t devProp;
  	hipGetDeviceProperties(&devProp, 0);
  	struct hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(LDPC_Sched_Stage_1_SCMS));
  	int nMP      = devProp.multiProcessorCount; // NOMBRE DE STREAM PROCESSOR
  	int nWarp    = attr.maxThreadsPerBlock/32;  // PACKET DE THREADs EXECUTABLES EN PARALLELE
  	int nThreads = nWarp * 32;					// NOMBRE DE THREAD MAXI PAR SP
  	int nDOF     = nb_frames;
  	int nBperMP  = 65536 / (attr.numRegs); 	// Nr of blocks per MP
  	int minB     = min(nBperMP*nThreads,1024);
  	int nBlocks  = max(minB/nThreads * nMP, nDOF/nThreads);  //Total number of blocks
  	printf("(II) Nombre de Warp    : %d\n", nWarp);
  	printf("(II) Nombre de Threads           : %d\n", nThreads);
  	printf("(II) LDPC_Sched_Stage_1_SCMS (PTX version %d)\n", attr.ptxVersion);
  	printf("(II) - Nombre de regist/thr : %d\n", attr.numRegs);
  	printf("(II) - Nombre de local/thr  : %ld\n", attr.localSizeBytes);
    printf("(II) - Nombre de shared/thr : %ld\n", attr.sharedSizeBytes);
    printf("(II) - Nombre de pBLOCKs    : %f\n", (float)nb_frames / (float)BLOCK_SIZE);
    printf("(II) - Nombre de pBLOCKs/uP : %f\n", (float)nb_frames / (float)BLOCK_SIZE / (float)devProp.multiProcessorCount);
}


CGPU_Decoder_SCMS::~CGPU_Decoder_SCMS()
{

}

void CGPU_Decoder_SCMS::initialize()
{

}

void CGPU_Decoder_SCMS::decode(float Intrinsic_fix[_N], int Rprime_fix[_N], int nombre_iterations)
{
    hipError_t Status;
	int nb_blocks = nb_frames / BLOCK_SIZE;
	Status = hipMemcpy/*Async*/(d_MSG_C_2_V, Intrinsic_fix, sz_nodes * sizeof(float), hipMemcpyHostToDevice);
    ERROR_CHECK(Status, __FILE__, __LINE__);

	{
    	// ORDERING THE LDPC CODEWORDS FOR DECODING (INTERLEAVING DATA)
		unsigned int NB_TRAMES    = _N;
		unsigned int FRAME_LENGTH = nb_frames;
		dim3 grid(NB_TRAMES/TILE_DIM, FRAME_LENGTH/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS);
		transposeDiagonal<<<grid, threads>>>((float*)device_V, (float*)d_MSG_C_2_V, _N, nb_frames);
	}

//	printf("processing decode (%d, %d, %d) !\n", nb_blocks, BLOCK_SIZE, nombre_iterations);
	LDPC_Sched_Stage_1_SCMS<<<nb_blocks, BLOCK_SIZE>>>(device_V, d_MSG_C_2_V, d_transpose, nombre_iterations);

	{
		// REORDERING THE LDPC CODEWORDS
		unsigned int NB_TRAMES    = nb_frames;
		unsigned int FRAME_LENGTH = _N;
		dim3 grid(NB_TRAMES/TILE_DIM, FRAME_LENGTH/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS);
		transposeDiagonal_and_hard_decision<<<grid, threads>>>((float*)d_MSG_C_2_V, (float*)device_V, NB_TRAMES, FRAME_LENGTH);
	}

    Status = hipMemcpy(Rprime_fix, d_MSG_C_2_V, sz_nodes * sizeof(float), hipMemcpyDeviceToHost);
	ERROR_CHECK(Status, __FILE__, __LINE__);
}
